#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

/* Utility function/macro, used to do error checking.
   Use this function/macro like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
*/
/*
#define checkCudaCall(result) {                                     \
    if (result != hipSuccess){                                     \
        cerr << "cuda error: " << hipGetErrorString(result);       \
        cerr << " in " << __FILE__ << " at line "<< __LINE__<<endl; \
        exit(1);                                                    \
    }                                                               \
}
*/
///*


static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}
//*/



__global__ void rgb2grayCudaKernel(unsigned char *d_inputImage, unsigned char *d_grayImage, int ImageSize)
{
        unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        
	if(index < ImageSize)

           {
                      float grayPix = 0.0f;
                      float r = static_cast< float >(d_inputImage[index]);
                      float g = static_cast< float >(d_inputImage[ImageSize + index]);
                      float b = static_cast< float >(d_inputImage[(2 * ImageSize) + index]);
                      grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);
                      d_grayImage[index] = static_cast< unsigned char >(grayPix);
        }
   }



void rgb2grayCuda(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height)
{

        NSTimer kernelTime = NSTimer("kernelTime", false, false);
        memset(reinterpret_cast< void * >(grayImage), 0, width * height * sizeof(unsigned char));

        unsigned char* d_grayImage= NULL;
        unsigned char* d_inputImage= NULL;
        int ImageSize = width * height;
        checkCudaCall(hipMalloc( (void **) &d_inputImage, (3*ImageSize) ));
        checkCudaCall(hipMalloc((void **) &d_grayImage, ImageSize));
        checkCudaCall(hipMemcpy(d_grayImage, grayImage, ImageSize, hipMemcpyHostToDevice));
        checkCudaCall(hipMemcpy(d_inputImage, inputImage, 3*ImageSize, hipMemcpyHostToDevice));

        kernelTime.start();
        dim3 dimBlock(512);
        dim3 dimGrid(ImageSize/(int)dimBlock.x);
        rgb2grayCudaKernel<<<dimGrid, dimBlock>>>(d_inputImage, d_grayImage, ImageSize);
        hipDeviceSynchronize();
        kernelTime.stop();

        checkCudaCall(hipMemcpy(grayImage, d_grayImage, ImageSize, hipMemcpyDeviceToHost));
        checkCudaCall(hipFree(d_grayImage));
        checkCudaCall(hipFree(d_inputImage));


        cout << fixed << setprecision(6);
        cout << "rgb2gray (gpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}

/*
void rgb2gray(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height) 
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	
	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			float grayPix = 0.0f;
			float r = static_cast< float >(inputImage[(y * width) + x]);
			float g = static_cast< float >(inputImage[(width * height) + (y * width) + x]);
			float b = static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);

			grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

			grayImage[(y * width) + x] = static_cast< unsigned char >(grayPix);
		}
	}
	// /Kernel
	kernelTime.stop();
	
	cout << fixed << setprecision(6);
	cout << "rgb2gray (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}
*/
/////////////////////////////////////

__global__ void histogram1DCudaKernel(int ImageSize, unsigned int *device_histogram, unsigned char *d_grayImage)
{
        unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < ImageSize)
        {
                unsigned char Item = d_grayImage[index];
                atomicAdd (&(device_histogram[Item]), 1);
        }
}



void histogram1DCuda(unsigned char *grayImage, unsigned char *histogramImage,const int width, const int height,
                                 unsigned int *histogram, const unsigned int HISTOGRAM_SIZE,
                                 const unsigned int BAR_WIDTH)
{
        dim3 threadBlockSize(512);
        unsigned int max = 0;
        int ImageSize = width * height;
        NSTimer kernelTime = NSTimer("kernelTime", false, false);

        memset(reinterpret_cast< void * >(histogram), 0, HISTOGRAM_SIZE * sizeof(unsigned int));
        unsigned int* device_histogram = NULL;
        checkCudaCall(hipMalloc((void **) &device_histogram, HISTOGRAM_SIZE* sizeof(unsigned int)));
        checkCudaCall(hipMemcpy(device_histogram, histogram, HISTOGRAM_SIZE* sizeof(unsigned int), hipMemcpyHostToDevice));
        unsigned char* d_grayImage = NULL;
        checkCudaCall(hipMalloc((void **) &d_grayImage, ImageSize));
        checkCudaCall(hipMemcpy(d_grayImage, grayImage, ImageSize, hipMemcpyHostToDevice));

        kernelTime.start();
        dim3 BlockNum(width*height/threadBlockSize.x+1);
        histogram1DCudaKernel<<<BlockNum, threadBlockSize>>>(ImageSize, device_histogram, d_grayImage);
        hipDeviceSynchronize();
        kernelTime.stop();

        checkCudaCall(hipMemcpy(histogram, device_histogram, HISTOGRAM_SIZE* sizeof(unsigned int), hipMemcpyDeviceToHost));
        checkCudaCall(hipFree(device_histogram));

	for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ )
	{
		if ( histogram[i] > max )
		{
			max = histogram[i];
		}
	}

	for ( int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH )
	{
		unsigned int value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( unsigned int y = 0; y < value; y++ )
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ )
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( unsigned int y = value; y < HISTOGRAM_SIZE; y++ )
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ )
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}

	cout << fixed << setprecision(6);
	cout << "histogram1D (gpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}

/*
void histogram1D(unsigned char *grayImage, unsigned char *histogramImage,const int width, const int height, 
				 unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
				 const unsigned int BAR_WIDTH) 
{
	unsigned int max = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	
	memset(reinterpret_cast< void * >(histogram), 0, HISTOGRAM_SIZE * sizeof(unsigned int));

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			histogram[static_cast< unsigned int >(grayImage[(y * width) + x])] += 1;
		}
	}
	// /Kernel
	kernelTime.stop();

	for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ ) 
	{
		if ( histogram[i] > max ) 
		{
			max = histogram[i];
		}
	}

	for ( int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH ) 
	{
		unsigned int value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( unsigned int y = 0; y < value; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( unsigned int y = value; y < HISTOGRAM_SIZE; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}
	
	cout << fixed << setprecision(6);
	cout << "histogram1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}
*/
/////////////////////////////////////
__global__ void contrast1DKernel(unsigned char *grayImage, const int width, const int height,int min, int max, int diff, int grayImageSize) 
{

  unsigned int index  = blockIdx.x * blockDim.x + threadIdx.x;

 
  //ensure we dont use more threads than image size 
  if(index < grayImageSize)
   {

   
	unsigned char pixel = grayImage[index];

         if ( pixel < min )
        {
        	pixel = 0;
        }
        else if ( pixel > max )
        {
        	pixel = 255;
        }
        else
        {
        	pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
        }
                      
        grayImage[index] =  pixel;
   
   } 
  

}

void contrast1DCuda(unsigned char *grayImage, const int width, const int height, 
				unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
				const unsigned int CONTRAST_THRESHOLD) 
{


      hipError_t error;

	unsigned int i = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	while ( (i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i++;
	}
	unsigned int min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i--;
	}
	unsigned int max = i;
	float diff = max - min;


	int threadBlockSize = 512;
       	int grayImageSize= width * height;
       

       	// Allocate device memory for grayImage
       	unsigned char *d_grayImage;
        checkCudaCall(hipMalloc((void **)&d_grayImage,grayImageSize));
       	
   	// Copy host memory to device 
         checkCudaCall(hipMemcpy(d_grayImage,grayImage,grayImageSize,hipMemcpyHostToDevice)); 
	
        // Setup execution parameters 
    	dim3 threads(512);
    	dim3 grid(grayImageSize/threads.x);


	kernelTime.start();
	// Kernel launch
	contrast1DKernel<<<grid,threads>>>(d_grayImage,width,height,min,max,diff,grayImageSize); 
    	hipDeviceSynchronize();
	kernelTime.stop();


        // Copy result from device to host 
        checkCudaCall(hipMemcpy(grayImage,d_grayImage,grayImageSize,hipMemcpyDeviceToHost));
	
	cout << fixed << setprecision(6);
	cout << "contrast1DCUDA (gpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
       
        // clean device memory 
        hipFree(d_grayImage); 

}

void contrast1D(unsigned char *grayImage, const int width, const int height, 
				unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
				const unsigned int CONTRAST_THRESHOLD) 
{
	unsigned int i = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	while ( (i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i++;
	}
	unsigned int min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i--;
	}
	unsigned int max = i;
	float diff = max - min;

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for (int x = 0; x < width; x++ ) 
		{
			unsigned char pixel = grayImage[(y * width) + x];

			if ( pixel < min ) 
			{
				pixel = 0;
			}
			else if ( pixel > max ) 
			{
				pixel = 255;
			}
			else 
			{
				pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
			}
			
			grayImage[(y * width) + x] = pixel;
		}
	}
	// /Kernel
	kernelTime.stop();
	
	cout << fixed << setprecision(6);
	cout << "contrast1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}

/////////////////////////////////////
/*
__global__ void triangularSmoothKernel
{
}
*/

/*
void triangularSmoothCuda
{
}
*/

void triangularSmooth(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height,
					  const float *filter) 
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	
	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			unsigned int filterItem = 0;
			float filterSum = 0.0f;
			float smoothPix = 0.0f;

			for ( int fy = y - 2; fy < y + 3; fy++ ) 
			{
				for ( int fx = x - 2; fx < x + 3; fx++ ) 
				{
					if ( ((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width)) ) 
					{
						filterItem++;
						continue;
					}

					smoothPix += grayImage[(fy * width) + fx] * filter[filterItem];
					filterSum += filter[filterItem];
					filterItem++;
				}
			}

			smoothPix /= filterSum;
			smoothImage[(y * width) + x] = static_cast< unsigned char >(smoothPix);
		}
	}
	// /Kernel
	kernelTime.stop();
	
	cout << fixed << setprecision(6);
	cout << "triangularSmooth (cpu): \t" << kernelTime.getElapsed() << " seconds." << endl;
}

