#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

/* Utility function/macro, used to do error checking.
   Use this function/macro like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));
   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
*/
/*
#define checkCudaCall(result) {                                     \
    if (result != hipSuccess){                                     \
        cerr << "cuda error: " << hipGetErrorString(result);       \
        cerr << " in " << __FILE__ << " at line "<< __LINE__<<endl; \
        exit(1);                                                    \
    }                                                               \
}
*/


#define BLOCK_MAX_THREADSIZE 512

//extern AppProfiler appProf;


extern double kernelCpuTime [4];
extern double kernelGpuTime [4];
extern double kernelSpeedUp [4];

//application profiling 
extern double AppCpuTime;
extern double AppGpuTime;
extern double AppSpeedUp;


static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}



__global__ void rgb2grayCudaKernel(unsigned char *d_inputImage, unsigned char *d_grayImage, int ImageSize)
{
        unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
        
	if(index < ImageSize)

           {
                      float grayPix = 0.0f;
                      float r = static_cast< float >(d_inputImage[index]);
                      float g = static_cast< float >(d_inputImage[ImageSize + index]);
                      float b = static_cast< float >(d_inputImage[(2 * ImageSize) + index]);
                      grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);
                      d_grayImage[index] = static_cast< unsigned char >(grayPix);
        }
   }



void rgb2grayCuda(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height)
{

        NSTimer kernelTime = NSTimer("kernelTime", false, false);
        memset(reinterpret_cast< void * >(grayImage), 0, width * height * sizeof(unsigned char));

        unsigned char* d_grayImage= NULL;
        unsigned char* d_inputImage= NULL;
        int ImageSize = width * height;
        checkCudaCall(hipMalloc( (void **) &d_inputImage, (3*ImageSize) ));
        checkCudaCall(hipMalloc((void **) &d_grayImage, ImageSize));
        checkCudaCall(hipMemcpy(d_grayImage, grayImage, ImageSize, hipMemcpyHostToDevice));
        checkCudaCall(hipMemcpy(d_inputImage, inputImage, 3*ImageSize, hipMemcpyHostToDevice));

        kernelTime.start();
        dim3 dimBlock(512);
        dim3 dimGrid(ImageSize/(int)dimBlock.x);
        rgb2grayCudaKernel<<<dimGrid, dimBlock>>>(d_inputImage, d_grayImage, ImageSize);
        hipDeviceSynchronize();
        kernelTime.stop();

        checkCudaCall(hipMemcpy(grayImage, d_grayImage, ImageSize, hipMemcpyDeviceToHost));
        checkCudaCall(hipFree(d_grayImage));
        checkCudaCall(hipFree(d_inputImage));


        cout << fixed << setprecision(6);
        cout << "rgb2gray (gpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	kernelGpuTime[0]= kernelTime.getElapsed(); 

}

void rgb2gray(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height) 
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	
	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			float grayPix = 0.0f;
			float r = static_cast< float >(inputImage[(y * width) + x]);
			float g = static_cast< float >(inputImage[(width * height) + (y * width) + x]);
			float b = static_cast< float >(inputImage[(2 * width * height) + (y * width) + x]);

			grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

			grayImage[(y * width) + x] = static_cast< unsigned char >(grayPix);
		}
	}
	// /Kernel
	kernelTime.stop();
	
	cout << fixed << setprecision(6);
	cout << "rgb2gray (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	kernelCpuTime[0]= kernelTime.getElapsed(); 
}

__global__ void histogram1DCudaKernel(int ImageSize, unsigned int *device_histogram, unsigned char *d_grayImage)
{
	// set the pointer to every element in d_grayImage
        unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < ImageSize)
        {
                unsigned char Item = d_grayImage[index];
                //use atomic operation to solve problem of memory conflict
                atomicAdd (&(device_histogram[Item]), 1);
        }
}



void histogram1DCuda(unsigned char *grayImage, unsigned char *histogramImage,const int width, const int height,
                                 unsigned int *histogram, const unsigned int HISTOGRAM_SIZE,
                                 const unsigned int BAR_WIDTH)
{
	// set the number of threads in a single block
        dim3 threadBlockSize(BLOCK_MAX_THREADSIZE);
        unsigned int max = 0;
        int ImageSize = width * height;
        NSTimer kernelTime = NSTimer("kernelTime", false, false);

        memset(reinterpret_cast< void * >(histogram), 0, HISTOGRAM_SIZE * sizeof(unsigned int));
        // copy histogram to device_histogram
        unsigned int* device_histogram = NULL;
        checkCudaCall(hipMalloc((void **) &device_histogram, HISTOGRAM_SIZE* sizeof(unsigned int)));
        checkCudaCall(hipMemcpy(device_histogram, histogram, HISTOGRAM_SIZE* sizeof(unsigned int), hipMemcpyHostToDevice));
        // copy grayImage to d_grayImage
        unsigned char* d_grayImage = NULL;
        checkCudaCall(hipMalloc((void **) &d_grayImage, ImageSize));
        checkCudaCall(hipMemcpy(d_grayImage, grayImage, ImageSize, hipMemcpyHostToDevice));

        kernelTime.start();
        // set the number of blocks
        dim3 BlockNum(width*height/threadBlockSize.x+1);
        histogram1DCudaKernel<<<BlockNum, threadBlockSize>>>(ImageSize, device_histogram, d_grayImage);
        hipDeviceSynchronize();
        kernelTime.stop();

        checkCudaCall(hipMemcpy(histogram, device_histogram, HISTOGRAM_SIZE* sizeof(unsigned int), hipMemcpyDeviceToHost));
        checkCudaCall(hipFree(device_histogram));
	// find the largest number in histogram
	for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ )
	{
		if ( histogram[i] > max )
		{
			max = histogram[i];
		}
	}

	for ( int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH )
	{
		unsigned int value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( unsigned int y = 0; y < value; y++ )
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ )
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( unsigned int y = value; y < HISTOGRAM_SIZE; y++ )
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ )
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}

	cout << fixed << setprecision(6);
	cout << "histogram1D (gpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	kernelGpuTime[1]= kernelTime.getElapsed(); 

}

void histogram1D(unsigned char *grayImage, unsigned char *histogramImage,const int width, const int height, 
				 unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
				 const unsigned int BAR_WIDTH) 
{
	unsigned int max = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	
	memset(reinterpret_cast< void * >(histogram), 0, HISTOGRAM_SIZE * sizeof(unsigned int));

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			histogram[static_cast< unsigned int >(grayImage[(y * width) + x])] += 1;
		}
	}
	// /Kernel
	kernelTime.stop();

	for ( unsigned int i = 0; i < HISTOGRAM_SIZE; i++ ) 
	{
		if ( histogram[i] > max ) 
		{
			max = histogram[i];
		}
	}

	for ( int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH ) 
	{
		unsigned int value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( unsigned int y = 0; y < value; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( unsigned int y = value; y < HISTOGRAM_SIZE; y++ ) 
		{
			for ( unsigned int i = 0; i < BAR_WIDTH; i++ ) 
			{
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}
	
	cout << fixed << setprecision(6);
	cout << "histogram1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	kernelCpuTime[1]= kernelTime.getElapsed(); 
}


__global__ void contrast1DKernel(unsigned char *grayImage, const int width, const int height,int min, int max, int diff, int grayImageSize) 
{

	unsigned int index  = blockIdx.x * blockDim.x + threadIdx.x;

  	//ensure we dont use more threads than image size 
  	if(index < grayImageSize)
   	{

		unsigned char pixel = grayImage[index];

         	if ( pixel < min )
        	{
        		pixel = 0;
        	}
        	else if ( pixel > max )
        	{
        		pixel = 255;
        	}
        	else
        	{
        		pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
        	}
                      
        	grayImage[index] =  pixel;
   
   	} 
  

}


void contrast1DCuda(unsigned char *grayImage, const int width, const int height, 
				unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
				const unsigned int CONTRAST_THRESHOLD) 
{

	unsigned int i = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	while ( (i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i++;
	}
	unsigned int min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i--;
	}
	unsigned int max = i;
	float diff = max - min;

       	int grayImageSize= width * height;
       
       	// Allocate device memory for grayImage
       	unsigned char *d_grayImage;
        checkCudaCall(hipMalloc((void **)&d_grayImage,grayImageSize));
       	
   	// Copy host memory to device 
         checkCudaCall(hipMemcpy(d_grayImage,grayImage,grayImageSize,hipMemcpyHostToDevice)); 
	
        // Setup execution parameters 
    	dim3 threads(BLOCK_MAX_THREADSIZE);
    	dim3 grid(grayImageSize/threads.x);

	// Kernel launch
	kernelTime.start();
	contrast1DKernel<<<grid,threads>>>(d_grayImage,width,height,min,max,diff,grayImageSize); 
    	hipDeviceSynchronize();
	kernelTime.stop();

        // Copy result from device to host 
        checkCudaCall(hipMemcpy(grayImage,d_grayImage,grayImageSize,hipMemcpyDeviceToHost));
	
	cout << fixed << setprecision(6);
	cout << "contrast1DCUDA (gpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	kernelGpuTime[2]= kernelTime.getElapsed(); 
       
        // clean device memory 
        hipFree(d_grayImage); 

}

void contrast1D(unsigned char *grayImage, const int width, const int height, 
				unsigned int *histogram, const unsigned int HISTOGRAM_SIZE, 
				const unsigned int CONTRAST_THRESHOLD) 
{
	unsigned int i = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	while ( (i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i++;
	}
	unsigned int min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) 
	{
		i--;
	}
	unsigned int max = i;
	float diff = max - min;

	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for (int x = 0; x < width; x++ ) 
		{
			unsigned char pixel = grayImage[(y * width) + x];

			if ( pixel < min ) 
			{
				pixel = 0;
			}
			else if ( pixel > max ) 
			{
				pixel = 255;
			}
			else 
			{
				pixel = static_cast< unsigned char >(255.0f * (pixel - min) / diff);
			}
			
			grayImage[(y * width) + x] = pixel;
		}
	}
	// /Kernel
	kernelTime.stop();
	
	cout << fixed << setprecision(6);
	cout << "contrast1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	kernelCpuTime[2]= kernelTime.getElapsed(); 
}

/////////////////////////////////////
/*
 * step 1. parallelize the triagularSmooth function by just adding the keyword __global__ in front of it
 * step 2. allocate the memory on the GPU and move the data over for the function to execute on
 * step 3. modify the function call in order to enable it to launch on the GPU
 */ 
 
 // 1. parallelize the triagularSmooth functionb by just adding the keyword __global__ in front of it
 // 
 // This function is called a Kernel: when called it is executed N times in parallel by N different
 // CUDA threads
 //
__global__ void triangularSmoothKernel(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height,
										const float *filter)
{
	// Each thread that executes the kernel is given a unique thread ID that is accessible within 
	// the kernel through the built-in threadIdx variable. 
	// Read more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#ixzz3nMa5Sr53
	
	// 'threadIdx' variable is a 3-component vector used to access the unique thread ID given to each 
	// thread that executes this Kernel
	// Each thread can be identified using a 1D, 2D or 3D thread index to form 1D, 2D or 3D block of threads

	// How many threads does a thread block on our GPU contains? 512 or 1024?
	
	// threadIdx:built-in variable used to access/identify a 1D, 2D or 3D thread index (and the thread-ID)
	// blockIdx: built-in variable used to access/identify a 1D, 2D or 3D block index
	// blockDim: built-in variable used to access a 1D, 2D or 3D thread dimension
	//
	// The total number of threads per block times the number of blocks
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; // read-only variable i
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; // read-only variable j
	
	if(i < width && j < height) /* ...??? */
	{
		unsigned int filterItem = 0;
		float filterSum = 0.0f;
		float smoothPix = 0.0f;

		for ( int fy = j - 2; fy < j + 3; fy++ ) 
		{
			for ( int fx = i - 2; fx < i + 3; fx++ ) 
			{
				if ( ((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width)) ) 
				{
					filterItem++;
					continue;
				}

				smoothPix += grayImage[(fy * width) + fx] * filter[filterItem];
				filterSum += filter[filterItem];
				filterItem++;
			}
		}

		smoothPix /= filterSum;
		smoothImage[(j * width) + i] = static_cast< unsigned char >(smoothPix);
	}
}

void triangularSmoothCuda(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height,
							const float *filter)
{
	//hipError_t error;
	
	//.....
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	
	//int threadBlockSize = 512;
	
	// 2a. Allocate the memory on the GPU
	unsigned char *d_grayImage;
	unsigned char *d_smoothImage;
	float *d_filter;
	checkCudaCall(hipMalloc((void **)&d_grayImage, width * height));
	checkCudaCall(hipMalloc((void **)&d_smoothImage, width * height));
	checkCudaCall(hipMalloc((void **)&d_filter, sizeof(filter)/sizeof(const float)));
	
	// 2b. Move data over (host memory to device memory) for the function to execute
	// hipMemcpy(void *dst, void *src, size_t nbytes, enum hipMemcpyKind direction); 
	checkCudaCall(hipMemcpy((void *)d_grayImage, (void *)grayImage, (hipMemcpyKind)width*height, hipMemcpyHostToDevice)); 
	checkCudaCall(hipMemcpy((void *)d_smoothImage, (void *)smoothImage, (hipMemcpyKind)width*height, hipMemcpyHostToDevice));
	checkCudaCall(hipMemcpy(d_filter, filter, (hipMemcpyKind)sizeof(filter)/sizeof(const float), hipMemcpyHostToDevice));	
	
	// 3. Modify the function call in order to enable it to launch on the GPU
	//dim3 threads(512);
	//dim3 grid((width * height)/threads.x);
	//
	// Kernel invocation with one block of width * height * 1 threads
	dim3 threadsPerBlock(16, 16);
	dim3 numberOfBlocks(width/threadsPerBlock.x, height/threadsPerBlock.y);
	
	kernelTime.start();
	//Kernel invocation with N threads that executes it
	// Kernel
	//triangularSmoothKernel<<<grid, threads>>>(d_grayImage, d_smoothImage, width, height, d_filter); 
	triangularSmoothKernel<<<numberOfBlocks, threadsPerBlock>>>(d_grayImage, d_smoothImage, width, height, d_filter); 
	hipDeviceSynchronize();	
	// /Kernel
	kernelTime.stop();
	
	// 4. Move data back over (device memory to host memory)
    checkCudaCall(hipMemcpy((void *)grayImage, (void *)d_grayImage, (hipMemcpyKind)width*height, hipMemcpyDeviceToHost));
	checkCudaCall(hipMemcpy((void *)smoothImage, (void *)d_smoothImage, (hipMemcpyKind)width*height, hipMemcpyDeviceToHost));
	
	cout << fixed << setprecision(6);
	cout << "triangularSmooth (gpu): \t" << kernelTime.getElapsed() << " seconds." << endl;
	
	kernelGpuTime[3]= kernelTime.getElapsed(); 
	// Free up device memory
	hipFree(d_grayImage); 
	hipFree(d_smoothImage); 
	hipFree(d_filter); 
}

void triangularSmooth(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height,
					  const float *filter) 
{
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	
	kernelTime.start();
	// Kernel
	for ( int y = 0; y < height; y++ ) 
	{
		for ( int x = 0; x < width; x++ ) 
		{
			unsigned int filterItem = 0;
			float filterSum = 0.0f;
			float smoothPix = 0.0f;

			for ( int fy = y - 2; fy < y + 3; fy++ ) 
			{
				for ( int fx = x - 2; fx < x + 3; fx++ ) 
				{
					if ( ((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width)) ) 
					{
						filterItem++;
						continue;
					}

					smoothPix += grayImage[(fy * width) + fx] * filter[filterItem];
					filterSum += filter[filterItem];
					filterItem++;
				}
			}

			smoothPix /= filterSum;
			smoothImage[(y * width) + x] = static_cast< unsigned char >(smoothPix);
		}
	}
	// /Kernel
	kernelTime.stop();
	
	cout << fixed << setprecision(6);
	cout << "triangularSmooth (cpu): \t" << kernelTime.getElapsed() << " seconds." << endl;
	kernelCpuTime[3]= kernelTime.getElapsed(); 
}

